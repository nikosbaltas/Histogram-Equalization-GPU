/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.005 

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

__global__ void RowGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR) {

  
  int k;
  int indexX = threadIdx.x + blockDim.x * blockIdx.x;
  int indexY = threadIdx.y + blockDim.y * blockIdx.y;
  int grid_width = gridDim.x * blockDim.x;
  int idx = indexY * grid_width + indexX;

  float sum = 0;
  for (k = -filterR; k <= filterR; k++) {
    int d = indexX + k;

    if (d >= 0 && d < imageW) {
      sum += d_Src[indexY * imageW + d] * d_Filter[filterR - k];
    }

  }
    d_Dst[idx] = sum;
}

__global__ void ColGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR) {
    
    
    int k;
    int indexX = threadIdx.x + blockDim.x * blockIdx.x;
    int indexY = threadIdx.y + blockDim.y * blockIdx.y;
    int grid_width = gridDim.x * blockDim.x;
    int idx = indexY * grid_width + indexX;


    float sum = 0;
    for (k = -filterR; k <= filterR; k++) {
      int d = indexY + k;

      if (d >= 0 && d < imageH) {
        sum += d_Src[d * imageW + indexX] * d_Filter[filterR - k];
      }
      
    }
    d_Dst[idx] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;

    float
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU;


    int imageW;
    int imageH;
    unsigned int i;

    struct timespec tv1, tv2;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);



    printf("Enter filter radius : ");
    scanf("%d", &filter_radius);

    //imageW = imageH = N

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));

    // Allocate memory for the device
    hipError_t mallocErr1 = hipMalloc((void **)&d_Filter, FILTER_LENGTH * sizeof(float));
    hipError_t mallocErr2 = hipMalloc((void **)&d_Input, imageW * imageH * sizeof(float));
    hipError_t mallocErr3 = hipMalloc((void **)&d_Buffer, imageW * imageH * sizeof(float));
    hipError_t mallocErr4 = hipMalloc((void **)&d_OutputGPU, imageW * imageH * sizeof(float));

    if (!h_Filter || !h_Input || !h_Buffer || !h_OutputCPU || !h_OutputGPU) {
      fprintf(stderr,"malloc error\n");
      exit(1);
    }

    if (mallocErr1 != hipSuccess || mallocErr2 != hipSuccess || 
        mallocErr3 != hipSuccess || mallocErr4 != hipSuccess ) {
      fprintf(stderr,"hipMalloc error\n");
      exit(1);
    }


    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }

    printf("CPU computation...\n");

    //Start of the CPU computation
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution rows
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution columns

    //End of the CPU computation
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);
    printf ("%g\n",
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec));
    


    dim3 grid_dim;
    dim3 block_dim;

    if (imageW > 32) {
      block_dim.x = 32;
      block_dim.y = 32;

      grid_dim.x = imageW / block_dim.x;
      grid_dim.y = imageH / block_dim.y;
    }
    else {
      grid_dim.x = 1;
      grid_dim.y = 1;

      block_dim.x = imageW;
      block_dim.y = imageH;
    }

    printf("GPU computation...\n");

    //Start measuring execution time of the two kernels
    hipEventRecord(start);

    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice);

    RowGPU<<< grid_dim, block_dim >>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    
    hipError_t err = hipGetLastError();

    if ( err != hipSuccess )
    {
        printf("CUDA Error1: %s\n", hipGetErrorString(err));       
        exit(-1);
    }

    hipDeviceSynchronize();

    ColGPU<<< grid_dim, block_dim>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius);
    
    if ( err != hipSuccess )
    {
        printf("CUDA Error2: %s\n", hipGetErrorString(err));       
        exit(-1);
    }

    
    hipDeviceSynchronize();
    
    hipMemcpy(h_OutputGPU, d_OutputGPU, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("kernel time in ms: %f\n", milliseconds);

    for (i = 0; i < imageW * imageH; i++) {
      if(ABS(h_OutputGPU[i]- h_OutputCPU[i]) >= accuracy) {
        printf("error\n");
        break;
      }
    }

    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    free(h_OutputGPU);

    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);
    // Do a device reset just in case...
    hipDeviceReset();

    return 0;
}
