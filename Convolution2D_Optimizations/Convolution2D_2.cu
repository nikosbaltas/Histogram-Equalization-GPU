/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005 

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

__global__ void RowGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR) {

  
  int k;
  int indexX = threadIdx.x + blockDim.x * blockIdx.x;
  int indexY = threadIdx.y + blockDim.y * blockIdx.y;
  int grid_width = gridDim.x * blockDim.x;
  int idx = indexY * grid_width + indexX;

  float sum = 0;
  for (k = -filterR; k <= filterR; k++) {
    int d = indexX + k;

    if (d >= 0 && d < imageW) {
      sum += d_Src[indexY * imageW + d] * d_Filter[filterR - k];
    }

  }
    d_Dst[idx] = sum;
}

__global__ void ColGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR) {
    
    
    int k;
    int indexX = threadIdx.x + blockDim.x * blockIdx.x;
    int indexY = threadIdx.y + blockDim.y * blockIdx.y;
    int grid_width = gridDim.x * blockDim.x;
    int idx = indexY * grid_width + indexX;


    float sum = 0;
    for (k = -filterR; k <= filterR; k++) {
      int d = indexY + k;

      if (d >= 0 && d < imageH) {
        sum += d_Src[d * imageW + indexX] * d_Filter[filterR - k];
      }

      
    }
    d_Dst[idx] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;

    float
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU;


    int imageW;
    int imageH;
    unsigned int i;


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("Enter filter radius : ");
    scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));

    // Allocate memory for the device
    hipError_t mallocErr1 = hipMalloc((void **)&d_Filter, FILTER_LENGTH * sizeof(float));
    hipError_t mallocErr2 = hipMalloc((void **)&d_Input, imageW * imageH * sizeof(float));
    hipError_t mallocErr3 = hipMalloc((void **)&d_Buffer, imageW * imageH * sizeof(float));
    hipError_t mallocErr4 = hipMalloc((void **)&d_OutputGPU, imageW * imageH * sizeof(float));


    if (!h_Filter || !h_Input || !h_Buffer || !h_OutputCPU || !h_OutputGPU) {
      fprintf(stderr,"malloc error\n");
      exit(1);
    }
    if (mallocErr1 != hipSuccess || mallocErr2 != hipSuccess || 
        mallocErr3 != hipSuccess || mallocErr4 != hipSuccess ) {
      fprintf(stderr,"hipMalloc error\n");
      exit(1);
    }

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }

    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice);
    

    printf("CPU computation...\n");

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution rows
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution columns



    dim3 grid_dim(1,1);
    dim3 block_dim(imageW,imageH);

    printf("GPU computation...\n");

    //Start measuring execution time of the two kernels
    hipEventRecord(start);

    RowGPU<<< grid_dim, block_dim>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();

    ColGPU<<< grid_dim, block_dim>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    
    hipEventRecord(stop);

    hipMemcpy(h_OutputGPU, d_OutputGPU, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost);

    
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("kernel time in ms: %f\n", milliseconds);


    for (i = 0; i < imageW * imageH; i++) {
      if(ABS(h_OutputGPU[i]- h_OutputCPU[i]) >= accuracy) {
        printf("error\n");
        break;
      }
    }

    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    free(h_OutputGPU);

    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);
    // Do a device reset just in case...
    hipDeviceReset();


    return 0;
}
