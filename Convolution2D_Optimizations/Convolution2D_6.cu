/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.000000005 

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

__global__ void RowGPU(double *d_Dst, double *d_Src, double *d_Filter, int imageW, int imageH, int filterR) {

  
  int k;
  int indexX = threadIdx.x + blockDim.x * blockIdx.x;
  int indexY = threadIdx.y + blockDim.y * blockIdx.y;
  int grid_width = gridDim.x * blockDim.x;
  int idx = indexY * grid_width + indexX;

  double sum = 0;
  for (k = -filterR; k <= filterR; k++) {
    int d = indexX + k;

    if (d >= 0 && d < imageW) {
      sum += d_Src[indexY * imageW + d] * d_Filter[filterR - k];
    }

  }
    d_Dst[idx] = sum;
}

__global__ void ColGPU(double *d_Dst, double *d_Src, double *d_Filter, int imageW, int imageH, int filterR) {
    
    
    int k;
    int indexX = threadIdx.x + blockDim.x * blockIdx.x;
    int indexY = threadIdx.y + blockDim.y * blockIdx.y;
    int grid_width = gridDim.x * blockDim.x;
    int idx = indexY * grid_width + indexX;


    double sum = 0;
    for (k = -filterR; k <= filterR; k++) {
      int d = indexY + k;

      if (d >= 0 && d < imageH) {
        sum += d_Src[d * imageW + indexX] * d_Filter[filterR - k];
      }
      
    }
    d_Dst[idx] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    double
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;

    double
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU;


    int imageW;
    int imageH;
    unsigned int i;

    struct timespec tv1, tv2;


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("Enter filter radius : ");
    scanf("%d", &filter_radius);

    //imageW = imageH = N
    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(imageW * imageH * sizeof(double));
    h_Buffer    = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputGPU = (double *)malloc(imageW * imageH * sizeof(double));

    // Allocate memory for the device
    hipError_t mallocErr1 = hipMalloc((void **)&d_Filter, FILTER_LENGTH * sizeof(double));
    hipError_t mallocErr2 = hipMalloc((void **)&d_Input, imageW * imageH * sizeof(double));
    hipError_t mallocErr3 = hipMalloc((void **)&d_Buffer, imageW * imageH * sizeof(double));
    hipError_t mallocErr4 = hipMalloc((void **)&d_OutputGPU, imageW * imageH * sizeof(double));

    if (!h_Filter || !h_Input || !h_Buffer || !h_OutputCPU || !h_OutputGPU) {
      fprintf(stderr,"malloc error\n");
      exit(1);
    }

    if (mallocErr1 != hipSuccess || mallocErr2 != hipSuccess || 
        mallocErr3 != hipSuccess || mallocErr4 != hipSuccess ) {
      fprintf(stderr,"hipMalloc error\n");
      exit(1);
    }

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
    }

    
    printf("CPU computation...\n");

    //Start of the CPU computation
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution rows
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution columns

    //End of the CPU computation
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);
    printf ("%g\n",
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec));
    



    dim3 grid_dim;
    dim3 block_dim;

    if (imageW > 32) {
      block_dim.x = 32;
      block_dim.y = 32;

      grid_dim.x = imageW / block_dim.x;
      grid_dim.y = imageH / block_dim.y;
    }
    else {
      grid_dim.x = 1;
      grid_dim.y = 1;

      block_dim.x = imageW;
      block_dim.y = imageH;
    }

    printf("GPU computation...\n");

    //Start measuring execution time of the two kernels
    hipEventRecord(start);

    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(double), hipMemcpyHostToDevice);

    RowGPU<<< grid_dim, block_dim >>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    
    hipError_t err = hipGetLastError();

    if ( err != hipSuccess )
    {
        printf("CUDA Error1: %s\n", hipGetErrorString(err));       
        exit(-1);
    }

    hipDeviceSynchronize();

    ColGPU<<< grid_dim, block_dim>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius);
    
    if ( err != hipSuccess )
    {
        printf("CUDA Error2: %s\n", hipGetErrorString(err));       
        exit(-1);
    }

    
    hipDeviceSynchronize();
    
    hipMemcpy(h_OutputGPU, d_OutputGPU, imageW * imageH * sizeof(double), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("kernel time in ms: %f\n", milliseconds);

    for (i = 0; i < imageW * imageH; i++) {
      if(ABS(h_OutputGPU[i]- h_OutputCPU[i]) >= accuracy) {
        printf("error\n");
        break;
      }
    }

    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    free(h_OutputGPU);

    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);
    // Do a device reset just in case...
    hipDeviceReset();


    return 0;
}
