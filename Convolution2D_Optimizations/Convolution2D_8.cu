/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.5 

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = filterR; y < imageH+filterR; y++) {
    for (x =filterR; x < imageH+filterR; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        sum += h_Src[y * (imageW+2*filterR) + d] * h_Filter[filterR - k];

      }
      h_Dst[y * (imageW+2*filterR) + x] = sum;
    }
    
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = filterR; y < imageH+filterR; y++) {
    for (x =filterR; x < imageH+filterR; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;
        sum += h_Src[d * (imageW+2*filterR) + x] * h_Filter[filterR - k];    
      }
      h_Dst[y * (imageW+2*filterR) + x] = sum;
    }
  }
    
}

__global__ void RowGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR) {

  
  int k;
  int indexX = threadIdx.x + blockDim.x * blockIdx.x;
  int indexY = threadIdx.y + blockDim.y * blockIdx.y;
  int grid_width = gridDim.x * blockDim.x;
  int idx = indexY * (grid_width+2*filterR) + indexX;
  int padding_offset= (imageW+2*filterR)*filterR +filterR;


  float sum = 0;
  for (k = -filterR; k <= filterR; k++) {
    int d = indexX + k;
    sum += d_Src[indexY*(imageW+2*filterR) + d + padding_offset] * d_Filter[filterR - k];
  }
  d_Dst[idx + padding_offset] = sum;

}

__global__ void ColGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR) {
    
    
    int k;
    int indexX = threadIdx.x + blockDim.x * blockIdx.x;
    int indexY = threadIdx.y + blockDim.y * blockIdx.y;
    int grid_width = gridDim.x * blockDim.x;
    int idx = indexY * (grid_width+2*filterR) + indexX;
    int padding_offset= (imageW+2*filterR)*filterR +filterR;


    float sum = 0;
    for (k = -filterR; k <= filterR; k++) {
      int d = indexY + k;
      sum += d_Src[d*(imageW + 2*filterR) + indexX + padding_offset] * d_Filter[filterR - k];
    }
    d_Dst[idx + padding_offset] = sum;

}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;

    float
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU;


    int imageW;
    int imageH;
    int padding;
    int dim_padding;
    unsigned int i;

    struct timespec tv1, tv2;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("Enter filter radius : ");
    scanf("%d", &filter_radius);

    padding= 2 * filter_radius;
    //imageW = imageH = N  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;
    dim_padding = imageW +padding;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(dim_padding * dim_padding * sizeof(float));
    h_Buffer    = (float *)malloc(dim_padding * dim_padding * sizeof(float));
    h_OutputCPU = (float *)malloc(dim_padding * dim_padding * sizeof(float));
    h_OutputGPU = (float *)malloc(dim_padding * dim_padding * sizeof(float));

    // Allocate memory for the device
    hipError_t mallocErr1 = hipMalloc((void **)&d_Filter, FILTER_LENGTH * sizeof(float));
    hipError_t mallocErr2 = hipMalloc((void **)&d_Input, dim_padding * dim_padding * sizeof(float));
    hipError_t mallocErr3 = hipMalloc((void **)&d_Buffer, dim_padding * dim_padding * sizeof(float));
    hipError_t mallocErr4 = hipMalloc((void **)&d_OutputGPU, dim_padding * dim_padding * sizeof(float));

    if (!h_Filter || !h_Input || !h_Buffer || !h_OutputCPU || !h_OutputGPU) {
      fprintf(stderr,"malloc error\n");
      exit(1);
    }

    if (mallocErr1 != hipSuccess || mallocErr2 != hipSuccess || 
        mallocErr3 != hipSuccess || mallocErr4 != hipSuccess ) {
      fprintf(stderr,"hipMalloc error\n");
      exit(1);
    }


    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    //Initialize padding with zeros
    for (int i = 0; i < dim_padding; i++) {
        for (int j = 0; j < dim_padding; j++) {
            if (i < filter_radius || i > imageW + filter_radius -1 || j < filter_radius || j > filter_radius + imageW - 1) {
                h_Input[i+j*dim_padding]=0;
                h_Buffer[i+j*dim_padding]=0;
                h_OutputCPU[i+j*dim_padding]=0;
            }
            else {
              h_Input[i+j*dim_padding] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
            }
        }
    }


    printf("CPU computation...\n");

    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution rows
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution columns
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);
    
    printf ("%g\n",
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec));


    
    dim3 grid_dim;
    dim3 block_dim;

    if (imageW > 32) {
      block_dim.x = 32;
      block_dim.y = 32;

      grid_dim.x = imageW / block_dim.x;
      grid_dim.y = imageH / block_dim.y;
    }
    else {
      grid_dim.x = 1;
      grid_dim.y = 1;

      block_dim.x = imageW;
      block_dim.y = imageH;
    }

    printf("GPU computation...\n");

    //Start measuring execution time of the two kernels
    hipEventRecord(start);

    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Input, h_Input, dim_padding * dim_padding * sizeof(float), hipMemcpyHostToDevice);

    RowGPU<<< grid_dim, block_dim >>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    
    hipError_t err = hipGetLastError();

    if ( err != hipSuccess )
    {
        printf("CUDA Error1: %s\n", hipGetErrorString(err));       
        exit(-1);
    }

    hipDeviceSynchronize();

    ColGPU<<< grid_dim, block_dim>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius);
    
    if ( err != hipSuccess )
    {
        printf("CUDA Error2: %s\n", hipGetErrorString(err));       
        exit(-1);
    }

    
    hipDeviceSynchronize();

    hipMemcpy(h_OutputGPU, d_OutputGPU, dim_padding * dim_padding * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("kernel time in ms: %f\n", milliseconds);


    for (i = 0; i < imageW * imageH; i++) {
      if(ABS(h_OutputGPU[i]- h_OutputCPU[i]) >= accuracy) {
        printf("error\n");
        break;
      }
    }

    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    free(h_OutputGPU);

    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);
    // Do a device reset just in case...
    hipDeviceReset();


    return 0;
}
