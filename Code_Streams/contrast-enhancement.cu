#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

PGM_IMG contrast_enhancement_g(PGM_IMG img_in, int *hist, float *cdf_d, int *lut, unsigned char *out)
{
    PGM_IMG result;

    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

    mempcpy(result.img, out, img_in.h * img_in.w * sizeof(unsigned char));

    return result;
}
