#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

void run_cpu_gray_test(PGM_IMG img_in, char *out_filename, int *bins, float *cdf_d, int *lut, unsigned char *out);

int main(int argc, char *argv[])
{
    PGM_IMG img_ibuf_g;
    PGM_IMG img_obuf;

    if (argc != 3)
    {
        printf("Run with input file name and output file name as arguments\n");
        exit(1);
    }

    unsigned char *input, *output;
    unsigned char *d_input, *d_output;

    int *d_bins;
    int *bins;

    float *cdf;
    float *d_cdf;

    int *lut;
    int *d_lut;

    unsigned char *input0, *input1;

    int N_bins = 256;
    float kernel_time = 0;

    img_ibuf_g = read_pgm(argv[1]);
    img_obuf.img = (unsigned char *)malloc(img_ibuf_g.w * img_ibuf_g.h * sizeof(unsigned char));
    img_obuf.h = img_ibuf_g.h;
    img_obuf.w = img_ibuf_g.w;

    int N = img_ibuf_g.h * img_ibuf_g.w;
    int div2 = N / 2;

    input0 = (unsigned char *)malloc(div2 * sizeof(unsigned char));
    input1 = (unsigned char *)malloc((N - div2) * sizeof(unsigned char));

    for (int i = 0; i < N; i++)
    {
        if (i < div2)
            input0[i] = img_ibuf_g.img[i];
        else
            input1[i] = img_ibuf_g.img[i];
    }

    hipEvent_t start_kernel, stop_kernel;
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);

    size_t bytes = img_ibuf_g.h * img_ibuf_g.w * sizeof(unsigned char);
    size_t bytes_bins = N_bins * sizeof(int);
    size_t bytes_bins_f = N_bins * sizeof(float);

    // Allocate host memory
    input = (unsigned char *)malloc(bytes);
    output = (unsigned char *)malloc(bytes);

    bins = (int *)malloc(bytes_bins);
    lut = (int *)malloc(bytes_bins);
    cdf = (float *)malloc(bytes_bins_f);

    hipMalloc((void **)&d_input, bytes);

    hipMalloc((void **)&d_bins, bytes_bins);

    hipMalloc((void **)&d_cdf, bytes_bins_f);

    hipMalloc((void **)&d_lut, bytes_bins);

    hipMalloc((void **)&d_output, bytes);

    printf("Running contrast enhancement for gray-scale images.\n\n");

    // Init input values
    for (int i = 0; i < N; i++)
    {
        input[i] = img_ibuf_g.img[i];
        output[i] = 0;
    }

    for (int i = 0; i < N_bins; i++)
    {
        bins[i] = 0;
        cdf[i] = 0.0;
        lut[i] = 0;
    }

    // Set grid dims
    int THREADS = 512;
    int gridDim = (N + 1) / THREADS;

    // briskoume min
    int min = 0;
    int i = 0;

    hipMemcpy(d_input, input, bytes, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bins, bins, bytes_bins, hipMemcpyHostToDevice);
    hipMemcpy(d_cdf, cdf, bytes_bins_f, hipMemcpyHostToDevice);
    hipMemcpy(d_lut, lut, bytes_bins, hipMemcpyHostToDevice);
    hipMemcpy(d_output, output, bytes, hipMemcpyHostToDevice);

    int *bins0;
    int *bins1;

    hipMalloc((void **)&bins0, 256 * sizeof(int));
    hipMalloc((void **)&bins1, 256 * sizeof(int));

    unsigned char
        *d_Input0,
        *d_Input1;

    // Create streams
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    // N * N * sizeof(unsigned char)/2 is half the image
    hipMalloc((void **)&d_Input0, div2 * sizeof(unsigned char));
    hipMalloc((void **)&d_Input1, (N - div2) * sizeof(unsigned char));

    hipEventRecord(start_kernel);

    hipMemcpyAsync(d_Input0, input, div2 * sizeof(unsigned char), hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(d_Input1, input + div2 * sizeof(unsigned char), (N - div2) * sizeof(unsigned char), hipMemcpyHostToDevice, stream1);

    // hipDeviceSynchronize();

    int gridDim0 = (div2 + 1) / THREADS;
    int gridDim1 = (N - div2 + 1) / THREADS;

    calcH<<<gridDim0, THREADS, 0, stream0>>>(d_Input0, bins0, div2);
    calcH<<<gridDim1, THREADS, 0, stream1>>>(d_Input1, bins1, N - div2);
    hipDeviceSynchronize();

    // Summing the two histograms
    addH<<<1, 256>>>(bins0, bins1, d_bins); // and for a grid bigger thatn 1 x 1
    hipDeviceSynchronize();

    createCDF<<<1, 256>>>(d_cdf, d_bins, N_bins);
    hipDeviceSynchronize();

    // Get calculated bins from device and cdf
    hipMemcpy(bins, d_bins, bytes_bins, hipMemcpyDeviceToHost);

    while (min == 0)
    {
        min = bins[i++];
    }

    calcLut<<<1, 256>>>(d_cdf, d_lut, min, N_bins, N - min);
    hipDeviceSynchronize();

    equalizeIMG<<<gridDim, THREADS>>>(d_input, d_output, d_lut, N);
    hipDeviceSynchronize();

    hipMemcpy(img_obuf.img, d_output, bytes, hipMemcpyDeviceToHost);

    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);

    hipEventElapsedTime(&kernel_time, start_kernel, stop_kernel);
    printf("kernel time in ms: %f  \n", kernel_time);

    // run_cpu_gray_test(img_ibuf_g, argv[2], bins, cdf, lut, output);
    write_pgm(img_obuf, argv[2]);
    free_pgm(img_obuf);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_bins);
    hipFree(d_cdf);
    hipFree(d_lut);
    free_pgm(img_ibuf_g);

    hipDeviceReset();
    return 0;
}

void run_cpu_gray_test(PGM_IMG img_in, char *out_filename, int *bins, float *cdf, int *lut, unsigned char *out)
{
    unsigned int timer = 0;
    PGM_IMG img_obuf;

    printf("\nStarting CPU processing...\n");
    img_obuf = contrast_enhancement_g(img_in, bins, cdf, lut, out);
    write_pgm(img_obuf, out_filename);
    free_pgm(img_obuf);
}

PGM_IMG read_pgm(const char *path)
{
    FILE *in_file;
    char sbuf[256];

    PGM_IMG result;
    int v_max; //, i;
    in_file = fopen(path, "r");
    if (in_file == NULL)
    {
        printf("Input file not found!\n");
        exit(1);
    }

    fscanf(in_file, "%s", sbuf); /*Skip the magic number*/
    fscanf(in_file, "%d", &result.w);
    fscanf(in_file, "%d", &result.h);
    fscanf(in_file, "%d\n", &v_max);
    printf("Image size: %d x %d\n", result.w, result.h);

    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

    fread(result.img, sizeof(unsigned char), result.w * result.h, in_file);
    fclose(in_file);

    return result;
}

void write_pgm(PGM_IMG img, const char *path)
{
    FILE *out_file;
    out_file = fopen(path, "wb");
    fprintf(out_file, "P5\n");
    fprintf(out_file, "%d %d\n255\n", img.w, img.h);
    fwrite(img.img, sizeof(unsigned char), img.w * img.h, out_file);
    fclose(out_file);
}

void free_pgm(PGM_IMG img)
{
    free(img.img);
}
